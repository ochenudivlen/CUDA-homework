﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cassert>
#include <chrono>
#include <thread>
#include <iomanip>
#include <random>

//Количество потоков в блоке
#define blockDim 2

//Количество блоков в сетке
#define blocksPerGrid 2

//Тип, который будут иметь элементы матриц
#define BASE_TYPE double

//Функция вычисления числа, которое больше a и кратно b
int toMultiple(int a, int b) 
{
    int mod = a % b;
    if (mod != 0) 
    {
        mod = b - mod;
        return a + mod;
    }
    return a;
}

__device__ void colon(BASE_TYPE* A, BASE_TYPE* I, const int Arows, const int Acols, const int n)
{
    BASE_TYPE ratio = 0;

    if (blockIdx.x != n)
    {
        if (A[n * Acols + n] != 0)
        {
            ratio = A[blockIdx.x * Acols + n] / A[n * Acols + n];
        }
        else
        {
            return;
        }

        A[blockIdx.x * Acols + threadIdx.x] -= A[n * Acols + threadIdx.x] * ratio;
        I[blockIdx.x * Acols + threadIdx.x] -= I[n * Acols + threadIdx.x] * ratio;
    }
}

//Функция получения обратной матрицы
__global__ void inverseMatrix(BASE_TYPE* A, BASE_TYPE* I, const int Arows, const int Acols)
{
    for (int i = 0; i < Arows; i++)
    {
        colon(A, I, Arows, Acols, i);
    }

    BASE_TYPE t = A[blockIdx.x * Acols + blockIdx.x];

    A[blockIdx.x * Acols + threadIdx.x] /= t;
    I[blockIdx.x * Acols + threadIdx.x] /= t;
}

int main()
{
    //start, stop - for Kernel time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // количество строк и столбцов матрицы
    int Arows = 1;
    int Acols = 1;

    Arows = toMultiple(Arows, blockDim);
    printf("Arows = %d\n", Arows);

    Acols = toMultiple(Acols, blockDim);
    printf("Acols = %d\n\n", Acols);

    //Проверка матрицы на квадратность
    if (Arows != Acols)
    {
        std::cout << "Matrix is not square" << std::endl;
        assert(Arows == Acols);
    }

    size_t Asize = Arows * Acols * sizeof(BASE_TYPE);

    BASE_TYPE* h_A = (BASE_TYPE*)malloc(Asize);
    BASE_TYPE* h_I = (BASE_TYPE*)malloc(Asize);
    BASE_TYPE* h_B = (BASE_TYPE*)malloc(Asize);
    BASE_TYPE* h_C = (BASE_TYPE*)malloc(Asize);

    std::random_device device;
    std::mt19937_64 engine(device());
    std::uniform_real_distribution<> distribution(0.0, 1.0);

    //Заполнение матрицы числами
    for (int i = 0; i < Arows * Acols; i++)
    {
        h_A[i] = int(distribution(engine) * 10 + 1);
    }

    for (int i = 0; i < Arows * Acols; i++)
    {
        h_I[i] = 0;
    }

    for (int i = 0; i < Arows; i++)
    {
        h_I[i * Arows + i] = 1;
    }

    std::cout << std::setprecision(3) << std::fixed;

    for (int i = 0; i < Arows; i++)
    {
        for (int j = 0; j < Acols; j++)
        {
            std::cout << h_A[i * Arows + j] << "\t";
        }

        std::cout << std::endl;
    }

    std::cout << std::endl;

    for (int i = 0; i < Arows; i++)
    {
        for (int j = 0; j < Acols; j++)
        {
            std::cout << h_I[i * Arows + j] << "\t";
        }

        std::cout << std::endl;
    }

    std::cout << std::endl;

    BASE_TYPE* d_A = NULL;
    hipMalloc((void**)&d_A, Asize);

    BASE_TYPE* d_I = NULL;
    hipMalloc((void**)&d_I, Asize);

    hipMemcpy(d_A, h_A, Asize, hipMemcpyHostToDevice);
    hipMemcpy(d_I, h_I, Asize, hipMemcpyHostToDevice);

    hipEventRecord(start, 0);

    inverseMatrix<<<blocksPerGrid, blockDim>>>(d_A, d_I, Arows, Acols);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float KernelTime;
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("KernelTime: %.2f milliseconds\n\n", KernelTime);

    hipMemcpy(h_B, d_A, Asize, hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_I, Asize, hipMemcpyDeviceToHost);

    for (int i = 0; i < Arows; i++)
    {
        for (int j = 0; j < Acols; j++)
        {
            std::cout << h_B[i * Arows + j] << "\t";
        }

        std::cout << std::endl;
    }

    std::cout << std::endl;

    for (int i = 0; i < Arows; i++)
    {
        for (int j = 0; j < Acols; j++)
        {
            std::cout << h_C[i * Arows + j] << "\t";
        }

        std::cout << std::endl;
    }

    std::cout << std::endl;

    free(h_A);
    free(h_I);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_I);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}